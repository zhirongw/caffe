#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void StepForward(const int nthreads, const Dtype* bottom_data,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    top_data[index] = Dtype(bottom_data[index] > 0);
  }
}
template <typename Dtype>
void StepLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = top[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  StepForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_data);
}

template <typename Dtype>
void StepLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom[0]->mutable_gpu_diff());
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(StepLayer);

}  // namespace caffe
